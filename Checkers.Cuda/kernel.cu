#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h> 
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "move.cuh"
#include "board.cuh"
#include "mctsnode.h"
#include "mcts.h"

#define CUDA_CALL(ans) { GpuAssert((ans), __FILE__, __LINE__, true); }
inline void GpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void RolloutGames(Board* rollout_boards, int* results, int size)
{
	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;


	for (long long ind = threadID; ind < size; ind += numThreads)
	{
		Board current_board = rollout_boards[ind];

		Player player = current_board.Rollout();
		results[ind] = player == Player::BLACK ? 1 : 0;
	}
}

__host__ Move* GetPossibleMovesFromInputParameters(int number_of_moves, char* possible_moves_array)
{
	Move *moves_to_fill = new Move[100];
	int ind = 1;
	for (int i = 0; i != number_of_moves; i++)
	{
		char beated_pieces_count = possible_moves_array[ind++];
		char *beated_pieces = new char[10];
		for (int j = 0; j != beated_pieces_count; j++)
		{
			beated_pieces[j] = possible_moves_array[ind++];
		}
		char old_position = possible_moves_array[ind++];
		char new_position = possible_moves_array[ind++];
		moves_to_fill[i] = Move(
			old_position,
			new_position,
			beated_pieces_count,
			beated_pieces
		);
	}
	return moves_to_fill;
}

__host__ void DeallocateMctsNode(MctsNode *node)
{
	for (int i = 0; i != node->children.size(); i++)
	{
		DeallocateMctsNode(node->children[i]);
	}
	//delete [] node->board.pieces;
	delete node;
}

extern "C" int __declspec(dllexport) __stdcall MakeMoveGpu
(
	char board_size,
	int current_player,			//0 - bia�y, 1 - czarny
	char* board,					//0 - puste, 1 - bia�y pion, 2 - bia�a dama, 3 - czarny pion, 4 - czarna dama
	char* possible_moves
)
{
	Player player = current_player == 0 ? Player::WHITE : Player::BLACK;	//gracz dla kt�rego wybierany jest optymalny ruch
	int
		number_of_mcts_iterations = 25,										//liczba iteracji wykonana przez algorytm MCTS
		possible_moves_count = possible_moves[0],							//liczba mo�liwych ruch�w spo�r�d kt�rych wybierany jest najlepszy
		block_size = 1024,													//rozmiar gridu z kt�rego gpu ma korzysta�
		grid_size = 1024,													//rozmiar bloku z kt�rego gpu ma korzysta� 
		*results_d,															//wska�nik na pami�� w GPU przechowuj�cy wyniki symulacji w danej iteracji
		*results;															//wska�nik na pami�� w CPU przechowuj�cy wyniki symulacji w danej iteracji
	Board
		start_board = Board(board_size, board, player),						//pocz�tkowy stan planszy
		*boards_d,															//wska�nik na pami�� w GPU przechowuj�cy plansze do symulacji
		*boards_to_rollout;													//wska�nik na pami�� w CPU przechowuj�cy plansze do symulacji
	Move* moves;															//lista mo�liwych do wykonania ruch�w
	std::vector<MctsNode*> rollout_vector;									//wektor przechowuj�cy elementy, dla kt�rych powinna zosta� wykonana symulacja dla GPU
	Mcts mcts_algorithm = Mcts();											//algorytm wybieraj�cy optymalny ruch

	moves = GetPossibleMovesFromInputParameters(possible_moves_count, possible_moves);

	mcts_algorithm.GenerateRoot(start_board, possible_moves_count, moves);
	for (int i = 0; i != possible_moves_count; i++)
		if (moves[i].beated_pieces_count > 0)
			delete[] moves[i].beated_pieces;
	delete[] moves;

	while (number_of_mcts_iterations--)
	{
		rollout_vector.clear();
		for (int i = 0; i != block_size * grid_size; i++)
		{
			MctsNode* node = mcts_algorithm.SelectNode(mcts_algorithm.root);
			if (node == 0 || node->visited_in_current_iteration)
				break;
			mcts_algorithm.BackpropagateSimulations(node);
			rollout_vector.push_back(node);
		}

		results = new int[rollout_vector.size()];
		boards_to_rollout = new Board[rollout_vector.size()];

		for (int i = 0; i != rollout_vector.size(); i++)
		{
			boards_to_rollout[i] = rollout_vector[i]->board;
		}
		CUDA_CALL(hipMalloc((void**)&boards_d, rollout_vector.size() * sizeof(Board)));
		CUDA_CALL(hipMalloc((void**)&results_d, rollout_vector.size() * sizeof(int)));
		CUDA_CALL(hipMemset(boards_d, 0, rollout_vector.size() * sizeof(int)));
		CUDA_CALL(hipMemcpy(boards_d, boards_to_rollout, rollout_vector.size() * sizeof(Board), hipMemcpyHostToDevice));

		//alokalcja dynamicznych tablic w klasach
		for (int i = 0; i < rollout_vector.size(); i++)
		{
			int* hostData;
			CUDA_CALL(hipMalloc((void**)&hostData, sizeof(int) * boards_to_rollout[i].size));
			CUDA_CALL(hipMemcpy(hostData, boards_to_rollout[i].pieces, sizeof(int) * boards_to_rollout[i].size, hipMemcpyHostToDevice));
			CUDA_CALL(hipMemcpy(&(boards_d[i].pieces), &hostData, sizeof(int*), hipMemcpyHostToDevice));
		}

		size_t size;
		CUDA_CALL(hipDeviceGetLimit(&size, hipLimitStackSize));
		CUDA_CALL(hipDeviceSetLimit(hipLimitStackSize, 4096));
		RolloutGames << <4, 256>> > (boards_d, results_d, rollout_vector.size());
		
		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipGetLastError());
		CUDA_CALL(hipMemcpy(results, results_d, sizeof(int) * rollout_vector.size(), hipMemcpyDeviceToHost));
		CUDA_CALL(hipFree(boards_d));
		CUDA_CALL(hipFree(results_d));
		mcts_algorithm.BackpropagateResults(rollout_vector, results);

		delete[] results;
		delete[] boards_to_rollout;
	}

	int best_move = mcts_algorithm.GetBestMove();
	DeallocateMctsNode(mcts_algorithm.root);
	return best_move;
}

