#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h> 
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>

#include "move.cuh"
#include "board.cuh"
#include "mctsnode.h"
#include "mcts.h"

#define CUDA_CALL(ans) { GpuAssert((ans), __FILE__, __LINE__, true); }
inline void GpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void SetupCurandKernel(hiprandState *state)
{
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(1234, threadID, 0, &state[threadID]);
}

__global__ void RolloutKernel(hiprandState *curand_state, Board* rollout_boards, int* results, Move* possible_moves_device, int size)
{
	const long numThreads = blockDim.x * gridDim.x;
	const long threadID = blockIdx.x * blockDim.x + threadIdx.x;

	for (long long ind = threadID; ind < size; ind += numThreads)
	{
		Board current_board = rollout_boards[ind];

		Player player = current_board.RolloutGpu(&curand_state[ind], possible_moves_device, ind);
		results[ind] = player == Player::BLACK ? 1 : 0;
	}
}

__host__ Move* GetPossibleMovesFromInputParameters(int number_of_moves, char* possible_moves_array)
{
	Move *moves_to_fill = new Move[100];
	int ind = 1;
	for (int i = 0; i != number_of_moves; i++)
	{
		char beated_pieces_count = possible_moves_array[ind++];
		char beated_pieces[10];
		for (int j = 0; j != beated_pieces_count; j++)
		{
			beated_pieces[j] = possible_moves_array[ind++];
		}
		char old_position = possible_moves_array[ind++];
		char new_position = possible_moves_array[ind++];
		moves_to_fill[i] = Move(
			old_position,
			new_position,
			beated_pieces_count,
			beated_pieces
		);
	}
	return moves_to_fill;
}

__host__ void DeallocateMctsNode(MctsNode *node)
{
	for (int i = 0; i != node->children.size(); i++)
	{
		DeallocateMctsNode(node->children[i]);
	}
	//delete [] node->board.pieces;
	delete node;
}

extern "C" int __declspec(dllexport) __stdcall MakeMoveGpu
(
	char board_size,
	int current_player,			//0 - bia�y, 1 - czarny
	char* board,					//0 - puste, 1 - bia�y pion, 2 - bia�a dama, 3 - czarny pion, 4 - czarna dama
	char* possible_moves
)
{
	Player player = current_player == 0 ? Player::WHITE : Player::BLACK;	//gracz dla kt�rego wybierany jest optymalny ruch
	int
		number_of_mcts_iterations = 50,										//liczba iteracji wykonana przez algorytm MCTS
		possible_moves_count = possible_moves[0],							//liczba mo�liwych ruch�w spo�r�d kt�rych wybierany jest najlepszy
		block_size = 225,													//rozmiar gridu z kt�rego gpu ma korzysta�
		grid_size = 225,														//rozmiar bloku z kt�rego gpu ma korzysta� 
		*results_d,															//wska�nik na pami�� w GPU przechowuj�cy wyniki symulacji w danej iteracji
		*results,															//wska�nik na pami�� w CPU przechowuj�cy wyniki symulacji w danej iteracji
		duplication_count = 1;												//parametr okre�laj�cy ile li�ci duplikowa� przy symulacji GPU

	Board
		start_board = Board(board_size, board, player),						//pocz�tkowy stan planszy
		*boards_d,															//wska�nik na pami�� w GPU przechowuj�cy plansze do symulacji
		*boards_to_rollout;													//wska�nik na pami�� w CPU przechowuj�cy plansze do symulacji
	Move
		*moves,																//lista mo�liwych do wykonania ruch�w
		*possible_moves_d;													//wska�nik na pami�� w GPU przechowuj�cy globaln� tablic� wszystkich mo�liwych ruch�w w danym threadzie
	hiprandState *state_d;													//do obliczania warto�ci pseudolosowych w kernelu
	std::vector<MctsNode*> rollout_vector;									//wektor przechowuj�cy elementy, dla kt�rych powinna zosta� wykonana symulacja dla GPU
	Mcts mcts_algorithm = Mcts();											//algorytm wybieraj�cy optymalny ruch

	moves = GetPossibleMovesFromInputParameters(possible_moves_count, possible_moves);

	mcts_algorithm.GenerateRoot(start_board, possible_moves_count, moves);

	delete[] moves;

	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipDeviceReset());
	CUDA_CALL(hipMalloc(&state_d, block_size * grid_size * sizeof(hiprandState)));
	SetupCurandKernel << <block_size, grid_size >> > (state_d);

	while (number_of_mcts_iterations--)
	{
		rollout_vector.clear();
		for (int i = 0; i * duplication_count < block_size * grid_size; i++)
		{
			MctsNode* node = mcts_algorithm.SelectNode(mcts_algorithm.root);
			if (node == 0 || node->visited_in_current_iteration)
				break;
			mcts_algorithm.BackpropagateSimulations(node, duplication_count);
			for (int j = 0; j != duplication_count; j++)
				rollout_vector.push_back(node);
		}

		results = new int[rollout_vector.size()];
		boards_to_rollout = new Board[rollout_vector.size()];

		for (int i = 0; i != rollout_vector.size(); i++)
		{
			boards_to_rollout[i] = rollout_vector[i]->board;
		}
		CUDA_CALL(hipMalloc((void**)&boards_d, rollout_vector.size() * sizeof(Board)));
		CUDA_CALL(hipMalloc((void**)&results_d, rollout_vector.size() * sizeof(int)));
		CUDA_CALL(hipMalloc((void**)&possible_moves_d, rollout_vector.size() * sizeof(Move) * 1000));
		CUDA_CALL(hipMemset(boards_d, 0, rollout_vector.size() * sizeof(int)));
		CUDA_CALL(hipMemcpy(boards_d, boards_to_rollout, rollout_vector.size() * sizeof(Board), hipMemcpyHostToDevice));

		//alokalcja dynamicznych tablic w klasach
		//for (int i = 0; i < rollout_vector.size(); i++)
		//{
		//	int* hostData;
		//	CUDA_CALL(hipMalloc((void**)&hostData, sizeof(int) * boards_to_rollout[i].size));
		//	CUDA_CALL(hipMemcpy(hostData, boards_to_rollout[i].pieces, sizeof(int) * boards_to_rollout[i].size, hipMemcpyHostToDevice));
		//	CUDA_CALL(hipMemcpy(&(boards_d[i].pieces), &hostData, sizeof(int*), hipMemcpyHostToDevice));
		//}
		hipDeviceProp_t prop;

		CUDA_CALL(hipGetDeviceProperties(&prop, 0));
		CUDA_CALL(hipDeviceSetLimit(hipLimitStackSize, 32000));
		RolloutKernel << <block_size, grid_size >> > (state_d, boards_d, results_d, possible_moves_d, rollout_vector.size());

		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipGetLastError());
		CUDA_CALL(hipMemcpy(results, results_d, sizeof(int) * rollout_vector.size(), hipMemcpyDeviceToHost));
		CUDA_CALL(hipFree(boards_d));
		CUDA_CALL(hipFree(results_d));
		CUDA_CALL(hipFree(possible_moves_d));
		mcts_algorithm.BackpropagateResults(rollout_vector, results);
		//for (int i = 0; i != rollout_vector.size(); i++)
		//{
		//	Player player = rollout_vector[i]->board.RolloutCpu();
		//}
		delete[] results;
		delete[] boards_to_rollout;
	}
	CUDA_CALL(hipFree(state_d));
	CUDA_CALL(hipDeviceReset());
	int best_move = mcts_algorithm.GetBestMove();
	DeallocateMctsNode(mcts_algorithm.root);
	return best_move;
}

