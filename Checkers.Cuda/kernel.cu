#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h> 
#include <vector>
//#include <cutil.h>		// timers
#include "board.h"
#include "Move.h"
#include "MCTS.h"

#define cConst 2

int N = 0;

std::vector<Move> GetPossibleMoves(int player, Board board)
{
	std::vector<Move> possibleMoves;
	return possibleMoves;
}

Board GetBoardAfterMove(Board board, Move move)
{
	int *pieces = new int[board.Size * board.Size];
	for (int i = 0; i != board.Size * board.Size; i++)
	{
		pieces[i] = board.Pieces[i];
	}
	for (int i = 0; i != move.BeatedPiecesCount; i++)
	{
		pieces[move.BeatedPieces[i]] = 0;
	}
	pieces[move.NewPosition] = pieces[move.OldPosition];
	pieces[move.OldPosition] = 0;
	return Board(board.Size, pieces);
}

MCTS* GenerateRoot(Board startBoard, int player, int movesCount, Move* possibleMoves)
{
	MCTS* root = new MCTS(NULL, startBoard, player);
	for (int i = 0; i != movesCount; i++)
	{
		MCTS* child = new MCTS(root, GetBoardAfterMove(startBoard, possibleMoves[i]), (player + 1) % 2);
		root->add_child(child);
	}
	return root;
}

MCTS* SelectNode(MCTS *parent)
{
	MCTS *leafNode = parent;
	while (leafNode->children.size() != 0)
	{
		int max = 0;
		int ind;
		for (int i = 0; i != leafNode->children.size(); i++)
		{
			if (leafNode->children[i]->simulationsCount == 0)
			{
				ind = i;
				break;
			}
			if (leafNode->children[i]->wins / leafNode->children[i]->simulationsCount + cConst * sqrt(log(N) / leafNode->children[i]->simulationsCount) > max)
			{
				max = leafNode->children[i]->wins / leafNode->children[i]->simulationsCount + cConst * sqrt(log(N) / leafNode->children[i]->simulationsCount);
				ind = i;
			}
		}
		leafNode = leafNode->children[ind];
	}
	if (leafNode->simulationsCount == 0)
	{
		return leafNode;
	}
	else
	{
		auto moves = GetPossibleMoves(leafNode->player, leafNode->board);
		if (moves.size() == 0)
			return NULL;
		for (int i = 0; i != moves.size(); i++)
		{
			leafNode->add_child(new MCTS(leafNode, GetBoardAfterMove(leafNode->board, moves[i]), (leafNode->player + 1) % 2));
		}

		return leafNode->children[0];
	}
}

void BackpropagateSimulations(MCTS *leaf)
{
	N++;
	while (leaf != NULL)
	{
		leaf->simulationsCount++;
		leaf = leaf->parent;
	}
}

__global__ void PredictNextMove(Board *board, Move* startingMoves)
{

}

// cuda kernel (internal)
__global__ void some_calculations(float *a, unsigned int N, unsigned int M)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		// note1: no need for shared memory here
		// note2: global memory access is coalesced
		//        (no structs, float only used)

		// do computations M times on each thread
		// to extend processor time
		for (unsigned int i = 0; i < M; i++)
		{
			// some easy arithmetics		
			a[idx] = a[idx] * a[idx] * 0.1 - a[idx] - 10;
		}
	}
}

// internal variable (example, not really necessary here)
static volatile int PRINT_ERRORS = 1;	// true

// cuda helper function (internal)
int checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		if (PRINT_ERRORS)
			printf("Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		return err;
	}
	return 0; // hipSuccess
}

extern "C" int __declspec(dllexport) __stdcall MakeMoveCpu
(
	int boardSize,
	int player, //0 - czrny, 1 - bia�y
	int* board, //0 - puste, 1 - bia�y pion, 2 - bia�a dama, 3 - czarny pion, 4 - czarna dama
	int* possibleMoves
)
{
	Board startBoard = Board(boardSize, board);
	int possibleMovesCount = possibleMoves[0];
	int ind = 1;
	Move* moves = new Move[possibleMovesCount];
	for (int i = 0; i != possibleMovesCount; i++)
	{
		int beatedPiecesCount = possibleMoves[ind++];
		int *beatedPieces = new int[beatedPiecesCount];
		for (int j = 0; j != beatedPiecesCount; j++)
		{
			beatedPieces[j] = possibleMoves[ind++];
		}
		moves[i] = Move(
			possibleMoves[ind++],
			possibleMoves[ind++],
			beatedPiecesCount,
			beatedPieces
		);
	}
	return possibleMovesCount - 1;
}

extern "C" int __declspec(dllexport) __stdcall MakeMoveGpu
(
	int boardSize,
	int player, //0 - bia�y, 1 - czarny
	int* board, //0 - puste, 1 - bia�y pion, 2 - bia�a dama, 3 - czarny pion, 4 - czarna dama
	int* possibleMoves
)
{
	Board startBoard = Board(boardSize, board);
	int possibleMovesCount = possibleMoves[0];
	int ind = 1;
	Move* moves = new Move[possibleMovesCount];
	for (int i = 0; i != possibleMovesCount; i++)
	{
		int beatedPiecesCount = possibleMoves[ind++];
		int *beatedPieces = new int[beatedPiecesCount];
		for (int j = 0; j != beatedPiecesCount; j++)
		{
			beatedPieces[j] = possibleMoves[ind++];
		}
		moves[i] = Move(
			possibleMoves[ind++],
			possibleMoves[ind++],
			beatedPiecesCount,
			beatedPieces
		);
	}

	MCTS* root = GenerateRoot(startBoard, player, possibleMovesCount, moves);
	std::vector<MCTS*> rolloutVector;
	for (int i = 0; i != 1000; i++)
	{
		MCTS* node = SelectNode(root);
		if (node == NULL || node->visitedInCurrentIteration)
			break;
		BackpropagateSimulations(node);
		rolloutVector.push_back(node);
	}
	int tmp = PRINT_ERRORS;
	int cuerr;
	int blockSize = 1024;      // The launch configurator returned block size 
	int gridSize = 1024;       // The actual grid size needed, based on input size 

	Move *moves_d;
	Board *board_d;

	hipMalloc((void**)&moves_d, possibleMovesCount * sizeof(Move));
	hipMalloc((void**)&board_d, sizeof(Board));
	hipMemcpy(moves_d, moves, possibleMovesCount * sizeof(Move), hipMemcpyHostToDevice);
	hipMemcpy(board_d, &startBoard, sizeof(Board), hipMemcpyHostToDevice);

	//alokalcja dynamicznych tablic w klasach
	int* hostData;
	hipMalloc((void**)&hostData, sizeof(int)*startBoard.Size);
	hipMemcpy(hostData, startBoard.Pieces, sizeof(int)*startBoard.Size, hipMemcpyHostToDevice);
	hipMemcpy(&(board_d->Pieces), &hostData, sizeof(int *), hipMemcpyHostToDevice);

	//alokalcja dynamicznych tablic w klasach
	for (int i = 0; i < possibleMovesCount; i++)
	{
		int* hostData;
		hipMalloc((void**)&hostData, sizeof(int)*moves[i].BeatedPiecesCount);
		hipMemcpy(hostData, moves[i].BeatedPieces, sizeof(int)*moves[i].BeatedPiecesCount, hipMemcpyHostToDevice);
		hipMemcpy(&(moves_d[i].BeatedPieces), &hostData, sizeof(int *), hipMemcpyHostToDevice);
	}

	//cutCreateTimer(&timer);			    // from cutil.h
	//cutStartTimer(timer);
	PredictNextMove << <gridSize, blockSize >> > (board_d, moves_d);	// kernel invocation
	hipDeviceSynchronize();			// by default kernel runs in parallel with CPU code
	//cutStopTimer(timer);

	cuerr = checkCUDAError("cuda kernel");

	//hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
	if (!cuerr) cuerr = checkCUDAError("cuda memcpy");

	//sExecutionTime = cutGetTimerValue(timer);

	hipFree(moves_d);
	if (!cuerr) cuerr = checkCUDAError("cuda free");

	PRINT_ERRORS = tmp;

	return possibleMovesCount - 1;
}
